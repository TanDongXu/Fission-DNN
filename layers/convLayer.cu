#include"convLayer.hpp"
#include"config/configBase.hpp"
#include"common/util/util.cuh"
#include"common/syncedmem.hpp"
#include"common/cudnn.hpp"
#include"common/common.hpp"
#include"test/test.hpp"

#include<hip/hip_runtime_api.h>
#include<glog/logging.h>

/*
 * Create handles
 */
template<typename Ntype>
void ConvLayer<Ntype>::createHandles()
{
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&bottom_tensorDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&top_tensorDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&biasTensorDesc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));
    hiprandCreateGenerator(&curandGenerator_W, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandCreateGenerator(&curandGenerator_B, HIPRAND_RNG_PSEUDO_MTGP32);
}

/*
 * Destroy the handles
 */
template<typename Ntype>
void ConvLayer<Ntype>:: destroyHandles()
{
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(bottom_tensorDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(top_tensorDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(biasTensorDesc));
    hiprandDestroyGenerator(curandGenerator_W);
   	hiprandDestroyGenerator(curandGenerator_B);
}

/*
 * Random initial weights and Bias
 */
template<typename Ntype>
void ConvLayer<Ntype>::initRandom(bool isGaussian)
{
    //srand((unsigned)time(NULL));
    if(isGaussian)
    {
        createGaussian<Ntype>(m_weight, m_epsilon);
    }else
    {
        //set seed
        hiprandSetPseudoRandomGeneratorSeed(curandGenerator_W, time(NULL));
        hiprandGenerateNormal(curandGenerator_W, (float*)m_weight->mutable_gpu_data(), m_kernelAmount * this->m_inputChannels * m_kernelSize * m_kernelSize, 0, m_epsilon);
    }
    // memset bias
    gpuMemoryMemset(m_bias->mutable_gpu_data(), m_kernelAmount * 1 * 1 * 1 * sizeof(Ntype));
}

template<typename Ntype>
void ConvLayer<Ntype>::ReShape()
{
    this->m_top = new NDMatrix<Ntype>(this->m_number, this->m_channels, this->m_height, this->m_width);
    m_weight = new NDMatrix<Ntype>(m_kernelAmount, this->m_inputChannels, m_kernelSize, m_kernelSize);
    m_bias = new NDMatrix<Ntype>(m_kernelAmount, 1, 1, 1);
}

/*
 * ConvLayer constructor
 */
template<typename Ntype>
ConvLayer<Ntype>::ConvLayer(string name)
{
    this->m_name = name;
    this->m_inputName = " ";
    this->m_bottom = NULL;
    this->m_top = NULL;
    this->m_loss = 0;
    this->m_prevLayer.clear();
    this->m_nextLayer.clear();
    m_weight = NULL;
    m_bias = NULL;
    tmp_Wgrad = NULL;
    tmp_Bgrad = NULL;

    filterDesc = NULL;
    convDesc = NULL;
    bottom_tensorDesc = NULL;
    top_tensorDesc = NULL;
    biasTensorDesc = NULL;
    convFwdAlgo = (hipdnnConvolutionFwdAlgo_t)-1;
    convBwdFilterAlgo = (hipdnnConvolutionBwdFilterAlgo_t)-1;
    convBwdDataAlgo = (hipdnnConvolutionBwdDataAlgo_t)-1;

    m_batchSize = ConfigTable::getInstance()->getBatchSize();
    m_momentum = ConfigTable::getInstance()->getMomentum();
    ConvLayerConfig* curConfig = (ConvLayerConfig*) ConfigTable::getInstance()->getLayerByName(this->m_name);
    bool isGaussian = curConfig->isGaussian();
    string prevLayerName = curConfig->getInput();
    Layer<Ntype>* prev_layer = (Layer<Ntype>*) LayerContainer<Ntype>::getInstance()->getLayerByName(prevLayerName);

    m_epsilon = curConfig->getInit_w();
    this->m_lrate = curConfig->getLrate();;
    m_kernelAmount = curConfig->getKernelAmount();
    m_kernelSize = curConfig->getKernelSize();
    m_pad_h = curConfig->getPad_h();
    m_pad_w = curConfig->getPad_w();
    m_stride_h = curConfig->getStride_h();
    m_stride_w = curConfig->getStride_w();
    m_lambda = curConfig->getWeight_decay();
    CHECK_EQ(m_pad_h, m_pad_w);
    CHECK_EQ(m_stride_h, m_stride_w);

    this->m_bottom = prev_layer->getTop();
    CHECK(this->m_bottom);
    this->m_inputChannels = this->m_bottom->ND_channels();
    m_prev_num = this->m_bottom->ND_num();
    m_prev_channels = this->m_inputChannels;
    m_prev_height = this->m_bottom->ND_height();
    m_prev_width = this->m_bottom->ND_width();
    this->m_number = m_prev_num;
    this->m_channels = m_kernelAmount;
    this->m_height = (m_prev_height + 2 * m_pad_h - m_kernelSize) / m_stride_h + 1;
    this->m_width = (m_prev_width + 2 * m_pad_w - m_kernelSize) / m_stride_w + 1;
    CHECK_EQ(this->m_height, this->m_width);
    CHECK_EQ(this->m_number, m_batchSize);
    
    mallocDeviceMem((void**)&tmp_Wgrad, this->m_inputChannels * m_kernelAmount * m_kernelSize * m_kernelSize * sizeof(float));
    mallocDeviceMem((void**)&tmp_Bgrad, m_kernelAmount * 1 * 1 * 1 * sizeof(float));
    // reShape the weight, bias, top , bottom
    this->createHandles();
    ReShape();
    this->initRandom(isGaussian);
}

///*
// * Deep copy constructor for convolution layers
// */
//ConvLayer::ConvLayer(const ConvLayer* layer)
//{
//    srcData = NULL;
//    dstData = NULL;
//    host_Weight = NULL;
//    host_Bias = NULL;
//    dev_Weight = NULL;
//    dev_Bias = NULL;
//    dev_Wgrad = NULL;
//    dev_Bgrad = NULL;
//    tmp_Wgrad = NULL;
//    tmp_Bgrad = NULL;
//    diffData = NULL;
//    prevLayer.clear();
//    nextLayer.clear();
//
//    filterDesc = NULL;
//    convDesc = NULL;
//    bottom_tensorDesc = NULL;
//    top_tensorDesc = NULL;
//    biasTensorDesc = NULL;
//    srcDiffTensorDesc = NULL;
//    dstDiffTensorDesc = NULL;
//    convFwdAlgo = (hipdnnConvolutionFwdAlgo_t)-1;
//    convBwdFilterAlgo = (hipdnnConvolutionBwdFilterAlgo_t)-1;
//    convBwdDataAlgo = (hipdnnConvolutionBwdDataAlgo_t)-1;
//
//    static int idx = 0;
//    _name = layer->_name + string("_") + int_to_string(idx);
//    idx ++;
//    _inputName = layer->_inputName ;
//    epsilon = layer->epsilon;
//    lrate = layer->lrate;
//    batchSize = layer->batchSize;
//    kernelAmount = layer->kernelAmount;
//    kernelSize = layer->kernelSize;
//    pad_h = layer->pad_h;
//    pad_w = layer->pad_w;
//    stride_h = layer->stride_h;
//    stride_w = layer->stride_w;
//    lambda = layer->lambda;
//    inputAmount = layer->inputAmount;
//    inputImageDim = layer->inputImageDim;
//    prev_num = layer->prev_num;
//    prev_channels = layer->prev_channels;
//    prev_height = layer->prev_height;
//    prev_width = layer->prev_width;
//    number = layer->number;
//    channels = layer->channels;
//    height = layer->height;
//    width = layer->width;
//    outputSize = layer->outputSize;
//
//    MemoryMonitor::getInstance()->gpuMallocMemory((void**)&dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
//    MemoryMonitor::getInstance()->gpuMallocMemory((void**)&dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
//    MemoryMonitor::getInstance()->gpuMallocMemory((void**)&tmp_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
//    MemoryMonitor::getInstance()->gpuMallocMemory((void**)&tmp_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
//    MemoryMonitor::getInstance()->gpuMallocMemory((void**)&dstData, batchSize * kernelAmount * height * width * sizeof(float));
//    MemoryMonitor::getInstance()->gpuMallocMemory((void**)&diffData, batchSize * inputAmount * inputImageDim * inputImageDim * sizeof(float));
//    //    MemoryMonitor::getInstance()->gpu2gpu(dev_Wgrad, layer->dev_Wgrad, kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
//    //    MemoryMonitor::getInstance()->gpu2gpu(dev_Bgrad, layer->dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
//    MemoryMonitor::getInstance()->gpuMemoryMemset(dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
//    MemoryMonitor::getInstance()->gpuMemoryMemset(dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
//    this->createHandles();
//    this->initRandom();
//    cout<<"Conv-copy"<<endl;
//}

/*
 * Destructor
 */
template<typename Ntype>
ConvLayer<Ntype>::~ConvLayer()
{
    destroyHandles();
    delete this->m_top;
    delete this->m_weight;
    delete this->m_bias;
}

/*
 * Forward propagation add Bias
 */
template<typename Ntype>
void ConvLayer<Ntype>::addBias(const hipdnnTensorDescriptor_t& top_tensorDesc, int c, Ntype* data )
{

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(biasTensorDesc,
                                          cuDNN<float>::getInstance()->GetTensorFormat(),
                                          cuDNN<float>::getInstance()->GetDataType(),
                                          1,
                                          c,
                                          1,
                                          1));

    float alpha = 1.0;
    float beta = 1.0;
    CUDNN_CHECK(hipdnnAddTensor(cuDNN<float>::getInstance()->GetcudnnHandle(),
                              &alpha,
                              biasTensorDesc,
                              m_bias->gpu_data(),
                              &beta,
                              top_tensorDesc,
                              data));
}

/*
 * Convolution forward propagation
 */
template<typename Ntype>
void ConvLayer<Ntype>::Forward(Phase phase)
{
    this->m_bottom = this->m_prevLayer[0]->getTop();

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(bottom_tensorDesc,
                                          cuDNN<float>::getInstance()->GetTensorFormat(),
                                          cuDNN<float>::getInstance()->GetDataType(),
                                          m_prev_num,
                                          m_prev_channels,
                                          m_prev_height,
                                          m_prev_width));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc,
                                          cuDNN<float>::getInstance()->GetDataType(),
                                          cuDNN<float>::getInstance()->GetTensorFormat(),
                                          m_kernelAmount,
                                          this->m_inputChannels,
                                          m_kernelSize,
                                          m_kernelSize));

    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(convDesc,
                                               m_pad_h,
                                               m_pad_w,//pading
                                               m_stride_h,
                                               m_stride_w,//stride
                                               1,1,//upscale
                                               HIPDNN_CROSS_CORRELATION));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(top_tensorDesc,
                                          cuDNN<float>::getInstance()->GetTensorFormat(),
                                          cuDNN<float>::getInstance()->GetDataType(),
                                          this->m_number,
                                          this->m_channels,
                                          this->m_height,
                                          this->m_width));

    /*
     * Obtain the best suited algorithm for cudnnConvolutinForward
     * */
    if (cuDNN<float>::getInstance()->getConvFwdAlgorithm() < 0)
    {
        CUDNN_CHECK(hipdnnGetConvolutionForwardAlgorithm(cuDNN<float>::getInstance()->GetcudnnHandle(),
                                                       bottom_tensorDesc,
                                                       filterDesc,
                                                       convDesc,
                                                       top_tensorDesc,
                                                       HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                       0,
                                                       &convFwdAlgo));

        cuDNN<float>::getInstance()->setConvolutionFwdAlgorithm(convFwdAlgo);
    }else
    {
    	convFwdAlgo =(hipdnnConvolutionFwdAlgo_t)cuDNN<float>::getInstance()->getConvFwdAlgorithm();
    }

    /*Get the amount of GPU memory for hipdnnConvolutionForward*/
    size_t convFwdSizeInBytes = 0;
    void* convFwdWorkSpace = NULL;
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cuDNN<float>::getInstance()->GetcudnnHandle(),
                                                       bottom_tensorDesc,
                                                       filterDesc,
                                                       convDesc,
                                                       top_tensorDesc,
                                                       convFwdAlgo,
                                                       &convFwdSizeInBytes));

    if (convFwdSizeInBytes != 0)
    {
        CUDA_CHECK(hipMalloc(&convFwdWorkSpace, convFwdSizeInBytes));
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionForward(cuDNN<float>::getInstance()->GetcudnnHandle(),
                                       &alpha,
                                       bottom_tensorDesc,
                                       this->m_bottom->gpu_data(),
                                       filterDesc,
                                       m_weight->gpu_data(),
                                       convDesc,
                                       convFwdAlgo,
                                       convFwdWorkSpace,
                                       convFwdSizeInBytes,
                                       &beta,
                                       top_tensorDesc,
                                       this->m_top->mutable_gpu_data()));

    /*add bias*/
    addBias(top_tensorDesc, this->m_channels,this->m_top->mutable_gpu_data());

    if (convFwdSizeInBytes != 0)
    {
        CUDA_CHECK(hipFree(convFwdWorkSpace));
    }

}

/*
 * Convolution backward propagation
 */
template<typename Ntype>
void ConvLayer<Ntype>::Backward()
{
    // Get the convolutuion function gradient with respect to the bias
    float alpha = 1.0f;
    float beta = 0.0f;
    //int nIndex = m_nCurBranchIndex;
    CUDNN_CHECK(hipdnnConvolutionBackwardBias(cuDNN<float>::getInstance()->GetcudnnHandle(),
                                            &alpha,
                                            top_tensorDesc,
                                            this->m_top->gpu_diff(),
                                            &beta,
                                            biasTensorDesc,
                                            tmp_Bgrad
                                            ));

    // Obtain the best suited algorithm for hipdnnConvolutionBackwardFilter
    if(cuDNN<float>::getInstance()->getConvolutionBwdFilterAlgorithm() < 0)
    {
    	CUDNN_CHECK(hipdnnGetConvolutionBackwardFilterAlgorithm(cuDNN<float>::getInstance()->GetcudnnHandle(),
    			                                               bottom_tensorDesc,
    			                                               top_tensorDesc,
    			                                               convDesc,
    			                                               filterDesc,
    			                                               HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
    			                                               0,
    			                                               &convBwdFilterAlgo
    			                                               ));

    	cuDNN<float>::getInstance()->setConvolutionBwdFilterAlgorithm(convBwdFilterAlgo);
    }else
    {
    	convBwdFilterAlgo = (hipdnnConvolutionBwdFilterAlgo_t)cuDNN<float>::getInstance()->getConvolutionBwdFilterAlgorithm();
    }

    /*Get the GPU memory workspace for hipdnnConvolutionBackwardFilter*/
    size_t convBwdFilterSizeInBytes = 0;
    void* convBwdFilterWorkSpace = NULL;
    CUDNN_CHECK(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cuDNN<float>::getInstance()->GetcudnnHandle(),
    		                                                  bottom_tensorDesc,
    		                                                  top_tensorDesc,
    		                                                  convDesc,
    		                                                  filterDesc,
    		                                                  convBwdFilterAlgo,
    		                                                  &convBwdFilterSizeInBytes
    /*Alloc GPU memory*/		                                                  ));
    if(convBwdFilterSizeInBytes != 0)
    {
    	CUDA_CHECK(hipMalloc(&convBwdFilterWorkSpace, convBwdFilterSizeInBytes));
    }

   /*This function computes the convolution gradient with respect to filter coefficient using the specified algo*/
    CUDNN_CHECK(hipdnnConvolutionBackwardFilter(cuDNN<float>::getInstance()->GetcudnnHandle(),
                                              &alpha,
                                              bottom_tensorDesc,
                                              this->m_bottom->gpu_data(),
                                              top_tensorDesc,
                                              this->m_top->gpu_diff(),
                                              convDesc,
                                              convBwdFilterAlgo,
                                              convBwdFilterWorkSpace,
                                              convBwdFilterSizeInBytes,
                                              &beta,
                                              filterDesc,
                                              tmp_Wgrad
                                              ));

    if (convBwdFilterSizeInBytes != 0)
    {
        CUDA_CHECK(hipFree(convBwdFilterWorkSpace));
    }

    /*Obtaining the best suited algorithm for the hipdnnConvolutionBackwardData*/
    if(cuDNN<float>::getInstance()->getConvolutionBwdDataAlgorithm() < 0)
    {
    	CUDNN_CHECK(hipdnnGetConvolutionBackwardDataAlgorithm(cuDNN<float>::getInstance()->GetcudnnHandle(),
    			                                            filterDesc,
    			                                            top_tensorDesc,
    			                                            convDesc,
    			                                            top_tensorDesc,
    			                                            HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
    			                                            0,
    			                                            &convBwdDataAlgo
    			                                            ));
    	cuDNN<float>::getInstance()->setConvolutionBwdDataAlgorithm(convBwdDataAlgo);

    }else
    {
    	convBwdDataAlgo = (hipdnnConvolutionBwdDataAlgo_t)cuDNN<float>::getInstance()->getConvolutionBwdDataAlgorithm();
    }

    /*Get the amount of GPU memory for the cudnnConvlotionBackwardData*/
    size_t convBwdDataSizeInBytes = 0;
    void* convBwdDataWorkSpace = NULL;
    /*按照接口说明bottom_tensorDesc应该是top_tensorDesc的,参考一个代码是用bottom_tensorDesc*/
    CUDNN_CHECK(hipdnnGetConvolutionBackwardDataWorkspaceSize(cuDNN<float>::getInstance()->GetcudnnHandle(),
    		                                                filterDesc,
    		                                                top_tensorDesc,
    		                                                convDesc,
    		                                                bottom_tensorDesc,
    		                                                convBwdDataAlgo,
    		                                                &convBwdDataSizeInBytes
    		                                                ));
    if(convBwdDataSizeInBytes != 0)
    {
    	CUDA_CHECK(hipMalloc(&convBwdDataWorkSpace, convBwdDataSizeInBytes));
    }

    //Note:if use convBwdDataAlgo above,it will return error in running.
    // convBwdDataAlgo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0;
    /*Compute the convolution gradient with respect to the output tensor using the specified algo*/
    alpha = 1.0f;
    beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionBackwardData(cuDNN<float>::getInstance()->GetcudnnHandle(),
                                            &alpha,
                                            filterDesc,
                                            m_weight->gpu_data(),
                                            top_tensorDesc,
                                            this->m_top->gpu_diff(),
                                            convDesc,
                                            convBwdDataAlgo,
                                            convBwdDataWorkSpace,
                                            convBwdDataSizeInBytes,
                                            &beta,
                                            bottom_tensorDesc,
                                            this->m_bottom->mutable_gpu_diff()));

    if(convBwdDataSizeInBytes != 0)
    {
    	CUDA_CHECK(hipFree(convBwdDataWorkSpace));
    }

    /*
     * Update the weights in conv layer
     *
     * */
    alpha = m_lambda * m_batchSize;
    int size =  m_kernelAmount * this->m_inputChannels * m_kernelSize * m_kernelSize;
    CUBLAS_CHECK(hipblasSaxpy(cuDNN<float>::getInstance()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  (float*)m_weight->mutable_gpu_data(),
                                  1,
                                  tmp_Wgrad,
                                  1));

    float scalVal = m_momentum;
    size =  m_kernelAmount * this->m_inputChannels * m_kernelSize * m_kernelSize;
    CUBLAS_CHECK(hipblasSscal(cuDNN<float>::getInstance()->GetcublasHandle(),
                                  size,
                                  &scalVal,
                                  (float*)m_weight->mutable_gpu_diff(),
                                  1));

    size = m_kernelAmount * 1 * 1 * 1;
    CUBLAS_CHECK(hipblasSscal(cuDNN<float>::getInstance()->GetcublasHandle(),
                                  size,
                                  &scalVal,
                                  (float*)m_bias->mutable_gpu_diff(),
                                  1));

    scalVal = this->m_lrate * 1.0f / m_batchSize;
    size =  m_kernelAmount * this->m_inputChannels * m_kernelSize * m_kernelSize;
    CUBLAS_CHECK(hipblasSaxpy(cuDNN<float>::getInstance()->GetcublasHandle(),
                                  size,
                                  &scalVal,
                                  tmp_Wgrad,
                                  1,
                                  (float*)m_weight->mutable_gpu_diff(),
                                  1));

    scalVal = 2 * this->m_lrate * 1.0f / m_batchSize;
    size = m_kernelAmount * 1 * 1 * 1;
    CUBLAS_CHECK(hipblasSaxpy(cuDNN<float>::getInstance()->GetcublasHandle(),
                                  size,
                                  &scalVal,
                                  tmp_Bgrad,
                                  1,
                                  (float*)m_bias->mutable_gpu_diff(),
                                  1));

    alpha = -1.0f;
    size =  m_kernelAmount * this->m_inputChannels * m_kernelSize * m_kernelSize;
    CUBLAS_CHECK(hipblasSaxpy(cuDNN<float>::getInstance()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  (float*)m_weight->mutable_gpu_diff(),
                                  1,
                                  (float*)m_weight->mutable_gpu_data(),
                                  1));

    size = m_kernelAmount * 1 * 1 * 1;
    CUBLAS_CHECK(hipblasSaxpy(cuDNN<float>::getInstance()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  (float*)m_bias->mutable_gpu_diff(),
                                  1,
                                  (float*)m_bias->mutable_gpu_data(),
                                  1));
}



INSTANTIATE_CLASS(ConvLayer);
